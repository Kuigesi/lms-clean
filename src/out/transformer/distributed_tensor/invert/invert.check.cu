#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif
#include "cudnn_header.h"
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include "cublas_header.h"
#include <stdbool.h>
#include "mpi_header.h"
#include "scanner_header.h"
/************* Functions **************/
__global__ void x10(float* x11, float x12, int x13) {
  // begin generating kernel function for FILL of type Float
  int x14 = gridDim.x * blockDim.x;
  int x15 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x15 < x13) {
    x11[x15] = x12;
    x15 = x15 + x14;
  }
  // end generating kernel function for FILL of type Float
}
__global__ void x19(float* x20, float* x21, int x22) {
  // begin generating kernel function for INVERT of type Float
  int x23 = gridDim.x * blockDim.x;
  int x24 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x24 < x22) {
    int x25 = x24;
    x21[x25] = 1.0 / x20[x25];
    x24 = x24 + x23;
  }
  // end generating kernel function for INVERT of type Float
}
__global__ void x27(float* x28, float* x29, float* x30, int x31) {
  // begin generating kernel function for ADD of type Float
  int x32 = gridDim.x * blockDim.x;
  int x33 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x33 < x31) {
    int x34 = x33;
    x30[x34] = x28[x34] + x29[x34];
    x33 = x33 + x32;
  }
  // end generating kernel function for ADD of type Float
}
__global__ void x39(float* x40, float* x41, int x42) {
  // begin generating kernel function for ACCUM of type Float
  int x43 = gridDim.x * blockDim.x;
  int x44 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x44 < x42) {
    int x45 = x44;
    x40[x45] = x40[x45] + x41[x45];
    x44 = x44 + x43;
  }
  // end generating kernel function for ACCUM of type Float
}
__global__ void x47(float* x48, float* x49, float* x50, int x51) {
  // begin generating kernel function for INVERT_GRAD of type Float
  int x52 = gridDim.x * blockDim.x;
  int x53 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x53 < x51) {
    int x54 = x53;
    float x55 = x48[x54];
    x50[x54] = 0.0 - x49[x54] / (x55 * x55);
    x53 = x53 + x52;
  }
  // end generating kernel function for INVERT_GRAD of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &x1));
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x3;
  NCCLCHECK(ncclGetUniqueId(&x3));
  MPICHECK(MPI_Bcast(&x3, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x4;
  NCCLCHECK(ncclCommInitRank(&x4, x1, x3, x2));
  hipStream_t x5;
  CUDA_CALL(hipStreamCreateWithFlags(&x5, hipStreamNonBlocking));
  int x6 = x2;
  // end setting up the MPI/NCCL environment
  // begin initializing GPU array of size 512 and type Float
  float* x7 = (float*)malloc(512 * sizeof(float));
  CUDA_CALL(hipSetDevice(x6));
  float* x8 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x8, (size_t)(512 * sizeof(float))));
  scan_float_array(x7, 512, "golden/weight_rank_%d.data", x6);
  CUDA_CALL(hipMemcpy(x8, x7, (size_t)(512 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing GPU array of size 512 and type Float
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x9 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x9, (size_t)(512 * sizeof(float))));
  x10<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x9, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  // begin initializing GPU array of size 512 and type Float
  float* x16 = (float*)malloc(512 * sizeof(float));
  CUDA_CALL(hipSetDevice(x6));
  float* x17 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x17, (size_t)(512 * sizeof(float))));
  scan_float_array(x16, 512, "golden/input_rank_%d.data", x6);
  CUDA_CALL(hipMemcpy(x17, x16, (size_t)(512 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing GPU array of size 512 and type Float
  // begin computing INV on GPU for size 512 and type Float at device (pre-rename) x39 with operand x45
  CUDA_CALL(hipSetDevice(x6));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(512 * sizeof(float))));
  x19<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x8, x18, 512);
  // end computing INV on GPU for size 512 and type Float at device (pre-rename) x39 with operand x45
  // begin computing ADD on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x103 and right_operand x120
  CUDA_CALL(hipSetDevice(x6));
  float* x26 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x26, (size_t)(512 * sizeof(float))));
  x27<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x17, x18, x26, 512);
  // end computing ADD on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x103 and right_operand x120
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x35 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x35, (size_t)(512 * sizeof(float))));
  x10<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x35, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x36 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x36, (size_t)(512 * sizeof(float))));
  x10<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x36, 0, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  // begin checking GPU array of size 512 and type Float
  float* x37 = (float*)malloc(512 * sizeof(float));
  CUDA_CALL(hipMemcpy(x37, x26, (size_t)(512 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array_with_file(x37, 512, "golden/loss_rank_%d.data", x6);
  // end checking GPU array of size 512 and type Float
  // begin initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x38 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x38, (size_t)(512 * sizeof(float))));
  x10<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x38, 1, 512);
  // end initializing fixed GPU array of size 512 and type Float and device (pre-rename) x39
  // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x219 and addition_operand x242
  CUDA_CALL(hipSetDevice(x6));
  x39<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x36, x38, 512);
  // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x219 and addition_operand x242
  // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x206 and addition_operand x242
  CUDA_CALL(hipSetDevice(x6));
  x39<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x35, x38, 512);
  // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x206 and addition_operand x242
  // begin computing INVERT_GRAD on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x45 and right_operand x219
  CUDA_CALL(hipSetDevice(x6));
  float* x46 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x46, (size_t)(512 * sizeof(float))));
  x47<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x8, x36, x46, 512);
  // end computing INVERT_GRAD on GPU for size 512 and type Float at device (pre-rename) x39 with left_operand x45 and right_operand x219
  // begin computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x62 and addition_operand x299
  CUDA_CALL(hipSetDevice(x6));
  x39<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x9, x46, 512);
  // end computing ACCUM on GPU for size 512 and type Float at device (pre-rename) x39 with base_operand x62 and addition_operand x299
  // begin checking GPU array of size 512 and type Float
  float* x56 = (float*)malloc(512 * sizeof(float));
  CUDA_CALL(hipMemcpy(x56, x9, (size_t)(512 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array_with_file(x56, 512, "golden/weight_grad_rank_%d.data", x6);
  // end checking GPU array of size 512 and type Float
  // begin checking GPU array of size 512 and type Float
  float* x57 = (float*)malloc(512 * sizeof(float));
  CUDA_CALL(hipMemcpy(x57, x35, (size_t)(512 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array_with_file(x57, 512, "golden/input_grad_rank_%d.data", x6);
  // end checking GPU array of size 512 and type Float
  NCCLCHECK(ncclCommDestroy(x4));
  MPICHECK(MPI_Finalize());
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
