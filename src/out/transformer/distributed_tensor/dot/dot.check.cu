#include "hip/hip_runtime.h"
/*****************************************
Emitting C Generated Code
*******************************************/
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif
#include "cudnn_header.h"
#include "nccl_header.h"
#include <string.h>
#include <cblas.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include "cublas_header.h"
#include <stdbool.h>
#include "mpi_header.h"
#include "scanner_header.h"
/************* Functions **************/
__global__ void x11(float* x12, float x13, int x14) {
  // begin generating kernel function for FILL of type Float
  int x15 = gridDim.x * blockDim.x;
  int x16 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x16 < x14) {
    x12[x16] = x13;
    x16 = x16 + x15;
  }
  // end generating kernel function for FILL of type Float
}
__global__ void x28(float* x29, float* x30, int x31) {
  // begin generating kernel function for ACCUM of type Float
  int x32 = gridDim.x * blockDim.x;
  int x33 = threadIdx.x + blockIdx.x * blockDim.x;
  while (x33 < x31) {
    int x34 = x33;
    x29[x34] = x29[x34] + x30[x34];
    x33 = x33 + x32;
  }
  // end generating kernel function for ACCUM of type Float
}
/**************** Snippet ****************/
void Snippet(int x0) {
  // begin setting up the MPI/NCCL environment
  int x1 = 0;
  int x2 = 0;
  MPICHECK(MPI_Init(NULL, NULL));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &x2));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &x1));
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipSetDevice(x2));
  ncclUniqueId x3;
  NCCLCHECK(ncclGetUniqueId(&x3));
  MPICHECK(MPI_Bcast(&x3, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, MPI_COMM_WORLD));
  ncclComm_t x4;
  NCCLCHECK(ncclCommInitRank(&x4, x1, x3, x2));
  hipStream_t x5;
  CUDA_CALL(hipStreamCreateWithFlags(&x5, hipStreamNonBlocking));
  int x6 = x2;
  // end setting up the MPI/NCCL environment
  hipblasHandle_t x7;
  CUBLAS_CALL(hipblasCreate(&x7));
  // begin initializing GPU array of size 208 and type Float
  float* x8 = (float*)malloc(208 * sizeof(float));
  CUDA_CALL(hipSetDevice(x6));
  float* x9 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x9, (size_t)(208 * sizeof(float))));
  scan_float_array(x8, 208, "golden/weight_rank_%d.data", x6);
  CUDA_CALL(hipMemcpy(x9, x8, (size_t)(208 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing GPU array of size 208 and type Float
  // begin initializing fixed GPU array of size 208 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x10 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x10, (size_t)(208 * sizeof(float))));
  x11<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, 0, 208);
  // end initializing fixed GPU array of size 208 and type Float and device (pre-rename) x39
  // begin initializing GPU array of size 208 and type Float
  float* x17 = (float*)malloc(208 * sizeof(float));
  CUDA_CALL(hipSetDevice(x6));
  float* x18 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x18, (size_t)(208 * sizeof(float))));
  scan_float_array(x17, 208, "golden/input_rank_%d.data", x6);
  CUDA_CALL(hipMemcpy(x18, x17, (size_t)(208 * sizeof(float)), hipMemcpyHostToDevice));
  // end initializing GPU array of size 208 and type Float
  // begin computing DOT on GPU for size 256 and type Float at device (pre-rename) x39 with left_operand x106 and right_operand x48
  CUDA_CALL(hipSetDevice(x6));
  float* x19 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x19, (size_t)(256 * sizeof(float))));
  float x20 = 1.0;
  float x21 = 0.0;
  CUBLAS_CALL(hipblasSgemm(x7, HIPBLAS_OP_N, HIPBLAS_OP_N, 16, 16, 13, &x20, x9, 16, x18, 13, &x21, x19, 16));
  // end computing DOT on GPU for size 256 and type Float at device (pre-rename) x39 with left_operand x106 and right_operand x48
  // begin initializing fixed GPU array of size 208 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x22 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x22, (size_t)(208 * sizeof(float))));
  x11<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x22, 0, 208);
  // end initializing fixed GPU array of size 208 and type Float and device (pre-rename) x39
  // begin checking GPU array of size 256 and type Float
  float* x23 = (float*)malloc(256 * sizeof(float));
  CUDA_CALL(hipMemcpy(x23, x19, (size_t)(256 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array_with_file(x23, 256, "golden/loss_rank_%d.data", x6);
  // end checking GPU array of size 256 and type Float
  // begin initializing fixed GPU array of size 256 and type Float and device (pre-rename) x39
  CUDA_CALL(hipSetDevice(x6));
  float* x24 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x24, (size_t)(256 * sizeof(float))));
  x11<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x24, 1, 256);
  // end initializing fixed GPU array of size 256 and type Float and device (pre-rename) x39
  // begin computing DOT on GPU for size 208 and type Float at device (pre-rename) x39 with left_operand x106 and right_operand x162 with transpose options
  CUDA_CALL(hipSetDevice(x6));
  float* x25 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x25, (size_t)(208 * sizeof(float))));
  float x26 = 1.0;
  float x27 = 0.0;
  CUBLAS_CALL(hipblasSgemm(x7, HIPBLAS_OP_N, HIPBLAS_OP_T, 16, 13, 16, &x26, x24, 16, x18, 13, &x27, x25, 16));
  // end computing DOT on GPU for size 208 and type Float at device (pre-rename) x39 with left_operand x106 and right_operand x162 with transpose options
  ncclAllReduce(x25, x25, (size_t)208, ncclFloat32, ncclSum, x4, x5);
  CUDA_CALL(hipStreamSynchronize(x5));
  // begin computing ACCUM on GPU for size 208 and type Float at device (pre-rename) x39 with base_operand x65 and addition_operand x175
  CUDA_CALL(hipSetDevice(x6));
  x28<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x10, x25, 208);
  // end computing ACCUM on GPU for size 208 and type Float at device (pre-rename) x39 with base_operand x65 and addition_operand x175
  // begin computing DOT on GPU for size 208 and type Float at device (pre-rename) x39 with left_operand x162 and right_operand x48 with transpose options
  CUDA_CALL(hipSetDevice(x6));
  float* x35 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x35, (size_t)(208 * sizeof(float))));
  float x36 = 1.0;
  float x37 = 0.0;
  CUBLAS_CALL(hipblasSgemm(x7, HIPBLAS_OP_T, HIPBLAS_OP_N, 13, 16, 16, &x36, x9, 16, x24, 16, &x37, x35, 13));
  // end computing DOT on GPU for size 208 and type Float at device (pre-rename) x39 with left_operand x162 and right_operand x48 with transpose options
  // begin computing ACCUM on GPU for size 208 and type Float at device (pre-rename) x39 with base_operand x139 and addition_operand x234
  CUDA_CALL(hipSetDevice(x6));
  x28<<<dim3(28, 1, 1), dim3(512, 1, 1)>>>(x22, x35, 208);
  // end computing ACCUM on GPU for size 208 and type Float at device (pre-rename) x39 with base_operand x139 and addition_operand x234
  // begin checking GPU array of size 208 and type Float
  float* x38 = (float*)malloc(208 * sizeof(float));
  CUDA_CALL(hipMemcpy(x38, x10, (size_t)(208 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array_with_file(x38, 208, "golden/weight_grad_rank_%d.data", x6);
  // end checking GPU array of size 208 and type Float
  // begin checking GPU array of size 208 and type Float
  float* x39 = (float*)malloc(208 * sizeof(float));
  CUDA_CALL(hipMemcpy(x39, x22, (size_t)(208 * sizeof(float)), hipMemcpyDeviceToHost));
  check_float_array_with_file(x39, 208, "golden/input_grad_rank_%d.data", x6);
  // end checking GPU array of size 208 and type Float
  NCCLCHECK(ncclCommDestroy(x4));
  CUBLAS_CALL(hipblasDestroy(x7));
  MPICHECK(MPI_Finalize());
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
